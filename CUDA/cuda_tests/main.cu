
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

using namespace std;

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


class CudaClass {
public:
    double *data;
    int a;

    CudaClass(double x) {
        data = new double[1];
        data[0] = x;
        a = x * 2;
    }
};

__global__ void useClass(CudaClass *cudaClass) {
    printf("%g, %d\n", cudaClass->data[0], cudaClass->a);
};

CudaClass *copyToGPU(CudaClass c) {
    // create class storage on device and copy top level class
    CudaClass *d_c;
    hipMalloc((void **) &d_c, sizeof(CudaClass));
    cudaCheckErrors("hipMalloc");
    hipMemcpy(d_c, &c, sizeof(CudaClass), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy");

    // make an allocated region on device for use by pointer in class
    double *hostdata;
    hipMalloc((void **) &hostdata, sizeof(double));
    cudaCheckErrors("hipMalloc");
    printf("%g\n", c.data[0]);
    hipMemcpy(hostdata, c.data, sizeof(double), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy");

    // copy pointer to allocated device storage to device class
    hipMemcpy(&(d_c->data), &hostdata, sizeof(double *), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy");
    return d_c;
}

int main() {
    CudaClass c(1);

    CudaClass *d_c = copyToGPU(c);

    useClass<<<1, 1>>>(d_c);
    hipDeviceSynchronize();
    return 0;
}