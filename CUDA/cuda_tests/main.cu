
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

using namespace std;

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


class CudaClass {
public:
    double *data;
    int a;

    CudaClass(double x) {
        data = new double[1];
        data[0] = x;
        a = x * 2;
    }
};

__global__ void useClass(CudaClass *cudaClass[]) {
    printf("kernel: \n");
    for (int i = 0; i < 10; i++) {
        printf("%d\n", i);
        printf("%g, %d\n", cudaClass[i]->data[0], cudaClass[i]->a);
    }
};

CudaClass *copyToGPU(CudaClass c) {
    // create class storage on device and copy top level class
    CudaClass *d_c;
    hipMalloc((void **) &d_c, sizeof(CudaClass));
    cudaCheckErrors("hipMalloc");
    hipMemcpy(d_c, &c, sizeof(CudaClass), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy");

    // make an allocated region on device for use by pointer in class
    double *hostdata;
    hipMalloc((void **) &hostdata, sizeof(double));
    cudaCheckErrors("hipMalloc");
    hipMemcpy(hostdata, c.data, sizeof(double), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy");

    // copy pointer to allocated device storage to device class
    hipMemcpy(&(d_c->data), &hostdata, sizeof(double *), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy");
    return d_c;
}

int main() {
    CudaClass *classes[10];

    for (int i = 0; i < 10; i++) {
        CudaClass c(i);
        classes[i] = copyToGPU(c);
    }

    // start kernel
    useClass<<<1, 1>>>(classes);
    hipDeviceSynchronize();
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));
    return 0;
}