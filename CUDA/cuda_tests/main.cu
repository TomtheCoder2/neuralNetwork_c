
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

using namespace std;

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


class CudaClass {
public:
    double *data;

    CudaClass(double x) {
        data = new double[1];
        data[0] = x;
    }
};

__global__ void useClass(CudaClass *cudaClass) {
    printf("%g\n", cudaClass->data[0]);
};

int main() {
    CudaClass c(1);

    // create class storage on device and copy top level class
    CudaClass *d_c;
    hipMalloc((void **) &d_c, sizeof(CudaClass));
    cudaCheckErrors("hipMalloc");
    hipMemcpy(d_c, &c, sizeof(CudaClass), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy");

    // make an allocated region on device for use by pointer in class
    double *hostdata;
    hipMalloc((void **) &hostdata, sizeof(double));
    cudaCheckErrors("hipMalloc");
    printf("%g\n", c.data[0]);
    hipMemcpy(hostdata, c.data, sizeof(double), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy");

    // copy pointer to allocated device storage to device class
    hipMemcpy(&(d_c->data), &hostdata, sizeof(double *), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy");

    useClass<<<1, 1>>>(d_c);
    hipDeviceSynchronize();
    return 0;
}